#include <stdio.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "malloc.h"

typedef struct counters
{
    int malloc_counter;
    int free_counter;
}counters_t;

__device__ void* linear_cudaMalloc(int size_in_bytes, counters_t *counter, void *g_base_addr)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0)
        hipMalloc((void**)g_base_addr, size_in_bytes);

    atomicSub(&counter->malloc_counter, 1);
    while(counter->malloc_counter > 0)
    {
        __syncthreads();
    }
    return (void*)(((*(char**)g_base_addr)) + (size_in_bytes * tid));
}

__device__ void linear_cudaFree(counters_t *counter, void *addr)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    atomicSub(&counter->free_counter, 1);
    while(counter->malloc_counter > 0);

    if(tid == 0)
    {
        hipFree(addr);
    }
}

__global__ void myKernel(int *output, int n)
{
    int *dev_array;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hipMalloc((void **)&dev_array, n * sizeof(int));

    for (int i = 0; i < n; i++)
    {
        dev_array[i] = (id * i);
    }

    for (int i = 0; i < n; i++)
    {
        output[id] += dev_array[i];
    }
    hipFree(dev_array);
}

__global__ void linear_malloc_kernel(int *output, int n, counters_t *counter, void *g_base_addr)
{
    int *dev_array;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    dev_array = (int*)linear_cudaMalloc(n*sizeof(int), counter, g_base_addr);

    printf("linear , %d , %p \n", id, dev_array);
    for (int i = 0; i < n; i++)
    {
        dev_array[i] = (id * i);
    }

    for (int i = 0; i < n; i++)
    {
        output[id] += dev_array[i];
    }
    linear_cudaFree(counter, dev_array);
}

int cpu_val(int max_tid, int malloc_size)
{
    int output = 0;
    for(int tid=0; tid<max_tid; tid++)
    {
        for (int i=0; i<malloc_size; i++)
        {
            output += (tid * i);
        }
    }
    return output;
}

// ./malloc blocks_per_grid, threads_per_block, malloc_size_per_thread_in_int_size
// ./malloc -g -b -n
int main(int argc, char **argv)
{
    int *dev_output;
    int grid_size, block_size, malloc_size;
    int opt;
    while ((opt = getopt(argc, argv, "g:b:n:")) != -1)
    {
        switch (opt)
        {
        case 'g':
            grid_size = atoi(optarg);
            break;
        case 'b':
            block_size = atoi(optarg);
            break;
        case 'n':
            malloc_size = atoi(optarg);
            break;
        case '?':
            printf("Usage: %s [-g grid_size] [-b block_size] [-n malloc_size_per_thread_in_int_size]\n", argv[0]);
            return 0;
        }
    }

    printf("%d, %d, %d\n", grid_size, block_size, malloc_size);
    int data_size = grid_size * block_size * sizeof(int);
    hipMallocManaged((void **)&dev_output, data_size);

    myKernel<<<grid_size, block_size>>>(dev_output, malloc_size);
    hipDeviceSynchronize();

    int output_sum = 0;
    for (int i = 0; i < data_size/sizeof(int); i++)
    {
        output_sum += dev_output[i];
    }
    int cpu_sum = cpu_val(grid_size * block_size, malloc_size);

    printf("gpu_output_sum %d, cpu_sum %d \n", output_sum, cpu_sum);

    if(output_sum != cpu_sum)
        printf("ERROR: CPU and GPU vals don't match!!!\n");
    else
        printf("default malloc-free test PASSED\n");

    counters_t *counter;
    void *g_base_address;
    hipMallocManaged((void**)&counter, sizeof(counters_t));

    counter->malloc_counter = grid_size * block_size;
    counter->free_counter = grid_size * block_size;

    hipMallocManaged(&g_base_address, sizeof(void*));

    hipMemset((void*)dev_output, 0, data_size);

    linear_malloc_kernel<<<grid_size, block_size>>>(dev_output, malloc_size, counter, g_base_address);
    hipDeviceSynchronize();

    output_sum = 0;
    for (int i = 0; i < data_size/sizeof(int); i++)
    {
        output_sum += dev_output[i];
    }
    printf("gpu_output_sum %d, cpu_sum %d \n", output_sum, cpu_sum);

    if(output_sum != cpu_sum)
        printf("ERROR: CPU and GPU vals don't match!!!\n");
    else
        printf("PASSED\n");

    hipFree(dev_output);

    return 0;
}